#include "hip/hip_runtime.h"
 // Headers
#include <cassert>
#include <iostream>
#include <vector>
#include <fstream>
#include <string>
#include <chrono>
#include <iomanip>
#include <stack>
#include <thread>
#include <algorithm>
#include <hip/hip_runtime.h>
#include <>
#include <sstream>
#include <cmath>

#include "gpuErrchk.h"

#define N_MAX 10 //max N board size

#define Threads 512 // threads allocation, hardware dependant (512 is the highest my hardware could go before failing)

// Struct to store result and total time taken
struct Result {
    int totalSolutions;
    std::chrono::duration<double> elapsedTime;
};

__device__ bool boardIsValidSoFar(int lastPlacedRow, const int* gameBoard, const int N)
{
    int lastPlacedColumn = gameBoard[lastPlacedRow];

    for (int row = 0; row < lastPlacedRow; ++row)
    {
        if (gameBoard[row] == lastPlacedColumn)
            return false;
        const auto col1 = lastPlacedColumn - (lastPlacedRow - row);
        const auto col2 = lastPlacedColumn + (lastPlacedRow - row);
        if (gameBoard[row] == col1 || gameBoard[row] == col2)
            return false;
    }
    return true;
}

__global__ void boardPossibilites(int N, long long int maxPossible, long long int offset, int* d_allSolutions, int* d_totalSolutions) {
    // Calculate the column index for the current thread
    long long int column = (long long int)(threadIdx.x + blockIdx.x * blockDim.x) + offset;
    if (column >= maxPossible)
        return;
    // Create an array to represent the game board
    int gameBoard[N_MAX];
    for (int i = 0; i < N; i++) {
        // Extract the column value for each row from the column index
        gameBoard[i] = column % N;
        // Check if the current placement is valid so far
        if (!boardIsValidSoFar(i, gameBoard, N))
            return;
        // Divide the column index by N to move to the next row
        column /= N;
    }
    // Increment the total solutions counter and get the new index
    const int index = atomicAdd(d_totalSolutions, 1);
    // Store the game board configuration for the current solution
    for (int i = 0; i < N; i++)
        d_allSolutions[N * index + i] = gameBoard[i];
}

Result systemStart(int N) {
    // Create a Result struct to store results
    Result result{}; 
    // Initialisation of total results amount
    result.totalSolutions = 0; 
    // Calculate the max number of possibilites 
    const long long int maxPossible = std::powl(N, N); 
    // Calculate the size of memory needed for solutions, N^5 to make sure there's any memory to store all solutions for higher N without getting a performance hit, 5 is highest it goes before slowdown on current hardware
    size_t solutionsMemory = std::powl(N, 5) * sizeof(int*); 
    // Pointer for the device memory to store all solutions and total solutions number
    int* d_allSolutions = 0; 
    int* d_totalSolutions = 0; 
    // Allocate device memory for all solutions and total solutions number
    hipMalloc((void**)&d_allSolutions, solutionsMemory);
    hipMalloc((void**)&d_totalSolutions, sizeof(int)); 
    // Number of kernel executions, set to one so that it runs
    int kernerExec = 1; 
    // Grid size for block execution set to double of thread size
    int grid = Threads * 2;
    // Block size for thread execution set to size of thread
    int block = Threads;
    // Check if the maximum possible number of combinations is greater than the grid * block size
    if (maxPossible > grid * block) 
        // Calculate the number of kernel executions needed
        kernerExec = std::ceil(static_cast<double>(maxPossible) / (grid * block)); 
    // Iterate over the number of kernel executions
    for (long long int i = 0; i < kernerExec; i++) { 
        // Launch the kernel with specified grid and block sizes
        boardPossibilites << <grid, block >> > (N, maxPossible, static_cast<long long int>(grid * block * i), d_allSolutions, d_totalSolutions); 
        // Synchronize device threads after kernel execution
        hipDeviceSynchronize(); 
    }
    // Copy the total number of solutions from device to host
    hipMemcpy(&result.totalSolutions, d_totalSolutions, sizeof(int), hipMemcpyDeviceToHost); 
    // Free device memory for the total number of solutions
    hipFree(d_totalSolutions);
    // Return the Result struct containing the total number of solutions
    return result; 
}

void calculateAllSolutions(int N) {
    Result result;
    auto start = std::chrono::system_clock::now();
    result = systemStart(N);
    auto stop = std::chrono::system_clock::now();

    result.elapsedTime = stop - start;

    printf("N=%d, solutions=%d\n", N, result.totalSolutions);
    printf("Time to find solutions for N=%d: %.0f microseconds\n", N, std::chrono::duration<double, std::micro>(result.elapsedTime).count());

}

int main(int argc, char** argv)
{
    gpuErrchk(hipSetDevice(0));

    for (int N = 4; N <= N_MAX; ++N)
        calculateAllSolutions(N);

    return 0;
}